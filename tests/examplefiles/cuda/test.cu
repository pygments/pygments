#include "hip/hip_runtime.h"
#include <stdio.h>

// __device__ function
__device__ void func()
{
    short* array0 = (short*)array;
    float* array1 = (float*)&array0[127];
}

/* __global__ function */
__global__ static void reduction(const float* __restrict__ input, float *output, clock_t *timer)
{
    // __shared__ float shared[2 * blockDim.x];
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    
    if (threadIdx.x == 0) {
        __threadfence();
    }

    // Perform reduction to find minimum.
    for (int d = blockDim.x; d > 0; d /= 2)
    {
        __syncthreads();
    }
}

int main(int argc, char **argv)
{
    dim3 dimBlock(8, 8, 1);

    timedReduction<<<dimBlock, 256, 256, 0>>>(dinput, doutput, dtimer);
    hipDeviceReset();
}
